#include "hip/hip_runtime.h"
#include "Raycaster.cuh"
#include "Kernels/Kernels.cuh"

void Raycaster::ChangeTree(CSGTree& tree)
{
    CleanUpTree();
    gpuErrchk(hipMalloc(&cudaTree.nodes, tree.nodes.size() * sizeof(CSGNode)));
    gpuErrchk(hipMalloc(&cudaTree.primitives, tree.primitives.primitives.size() * sizeof(Primitive)));

    gpuErrchk(hipMemcpy(cudaTree.nodes, tree.nodes.data(), tree.nodes.size() * sizeof(CSGNode), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitives, tree.primitives.primitives.data(), tree.primitives.primitives.size() * sizeof(Primitive), hipMemcpyHostToDevice));
    allocedTree = true;
}

void Raycaster::ChangeSize(int newWidth, int newHeight)
{
    CleanUpTexture();
    width = newWidth;
    height = newHeight;
    blockDim = dim3(BLOCKXSIZE, BLOCKYSIZE);
    gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    gpuErrchk(hipMalloc(&devHits, width * height * sizeof(RayHit)));
    alloced = true;
}

void Raycaster::ChangeSize(int newWidth, int newHeight, CSGTree& tree)
{
    ChangeSize(newWidth, newHeight);
    ChangeTree(tree);
}

void Raycaster::Raycast(float4* devPBO, Camera cam, DirectionalLight light)
{
    MapFromCamera(cam);

    if (alg == 0)
        RaycastKernel << <gridDim, blockDim >> > (cudaCamera, cudaTree, devHits, width, height);
    else if (alg == 1)
        CalculateInterscetion << <gridDim, blockDim >> > (width, height, shapeCount, cudaTree, devParts, cudaCamera, devHits);
    hipDeviceSynchronize();



    LightningKernel << <gridDim, blockDim >> > (cudaCamera, devHits, cudaTree.primitives, devPBO, light.getLightDir(), width, height);
    hipDeviceSynchronize();

}

void Raycaster::CleanUpTree()
{
    if (allocedTree)
    {
        gpuErrchk(hipFree(cudaTree.nodes));
        gpuErrchk(hipFree(cudaTree.primitives));
        allocedTree = false;
    }
}
void Raycaster::CleanUpTexture()
{
    if (alloced)
    {
        gpuErrchk(hipFree(devHits));
        alloced = false;
    }
}

void Raycaster::CleanUpClassical()
{
    if (allocedClassicalAdds)
    {
        gpuErrchk(hipFree(devParts));
        free(Parts);
        allocedClassicalAdds = false;
    }

}

void  Raycaster::SetupClassical(CSGTree& tree)
{
    if (allocedTree)
    {
        Parts = (int*)malloc(tree.primitives.primitives.size() * 4 * sizeof(int));
        CreateParts(tree, Parts, 0);
        gpuErrchk(hipMalloc(&devParts, tree.primitives.primitives.size() * 4 * sizeof(int)));
        gpuErrchk(hipMemcpy(devParts, Parts, tree.primitives.primitives.size() * 4 * sizeof(int), hipMemcpyHostToDevice));
        shapeCount = tree.primitives.primitives.size();
        allocedClassicalAdds = true;
    }
}

void Raycaster::CleanUp()
{
    CleanUpTree();
    CleanUpTexture();
    CleanUpClassical();
}

void Raycaster::ChangeAlg(CSGTree& tree, int newAlg)
{

    if (alg == 1)
        CleanUpClassical();
    ChangeTree(tree);
    alg = newAlg;
    if (alg == 1)
        SetupClassical(tree);

}

void Raycaster::MapFromCamera(Camera cam)
{
    cudaCamera.position = make_float3(cam.x, cam.y, cam.z);
    cudaCamera.forward = make_float3(cam.forward[0], cam.forward[1], cam.forward[2]);
    cudaCamera.right = make_float3(cam.right[0], cam.right[1], cam.right[2]);
    cudaCamera.up = make_float3(cam.up[0], cam.up[1], cam.up[2]);
    cudaCamera.fov = cam.fov;
}