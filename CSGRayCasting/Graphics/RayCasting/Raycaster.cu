#include "hip/hip_runtime.h"
#include "Raycaster.cuh"
#include "Kernels/Kernels.cuh"
#include "hip/hip_runtime_api.h"


void Raycaster::ChangeTree(CSGTree& tree)
{
    CleanUpTree();
    gpuErrchk(hipMalloc(&cudaTree.nodes, tree.nodes.size() * sizeof(CSGNode)));
    gpuErrchk(hipMalloc(&cudaTree.primitivePos, tree.primitives.primitivePos.size() * sizeof(CudaPrimitivePos)));
    gpuErrchk(hipMalloc(&cudaTree.primitiveColor, tree.primitives.primitiveColor.size() * sizeof(CudaPrimitiveColor)));
    gpuErrchk(hipMalloc(&cudaTree.primitiveParams, tree.primitives.primitiveParameters.size() * sizeof(Parameters)));

    gpuErrchk(hipMemcpy(cudaTree.nodes, tree.nodes.data(), tree.nodes.size() * sizeof(CSGNode), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitivePos, tree.primitives.primitivePos.data(), tree.primitives.primitivePos.size() * sizeof(CudaPrimitivePos), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitiveColor, tree.primitives.primitiveColor.data(), tree.primitives.primitiveColor.size() * sizeof(CudaPrimitiveColor), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitiveParams, tree.primitives.primitiveParameters.data(), tree.primitives.primitiveParameters.size() * sizeof(Parameters), hipMemcpyHostToDevice));
    allocedTree = true;
    nodeCount = tree.nodes.size();
}

void Raycaster::ChangeSize(int newWidth, int newHeight)
{
    CleanUpTexture();
    width = newWidth;
    height = newHeight;
    gpuErrchk(hipMalloc(&devHits, width * height * sizeof(RayHit)));
    alloced = true;
}

void Raycaster::ChangeSize(int newWidth, int newHeight, CSGTree& tree)
{
    ChangeSize(newWidth, newHeight);
    ChangeTree(tree);
}

void Raycaster::Raycast(float4* devPBO, Camera cam, DirectionalLight light)
{
    MapFromCamera(cam);
    hipProfilerStart();
    if (alg == 0)
        RaycastKernel << <gridDim, blockDim >> > (cudaCamera, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devHits, width, height);
    else if (alg == 1)
        CalculateInterscetion << <gridDim, blockDim >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
    else if (alg == 2)
        RaymarchingKernel << <gridDim, blockDim >> > (cudaCamera, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devHits, nodeCount, width, height);
    hipDeviceSynchronize();



    LightningKernel << <gridDim, blockDim >> > (cudaCamera, devHits, cudaTree.primitiveColor, devPBO, light.getLightDir(), width, height);
    hipDeviceSynchronize();
    hipProfilerStop();
}

void Raycaster::CleanUpTree()
{
    if (allocedTree)
    {
        gpuErrchk(hipFree(cudaTree.nodes));
        gpuErrchk(hipFree(cudaTree.primitivePos));
        gpuErrchk(hipFree(cudaTree.primitiveColor));
        gpuErrchk(hipFree(cudaTree.primitiveParams));
        allocedTree = false;
    }
}
void Raycaster::CleanUpTexture()
{
    if (alloced)
    {
        gpuErrchk(hipFree(devHits));
        alloced = false;
    }
}

void Raycaster::CleanUpClassical()
{
    if (allocedClassicalAdds)
    {
        gpuErrchk(hipFree(devParts));
        free(Parts);
        allocedClassicalAdds = false;
    }

}

void  Raycaster::SetupClassical(CSGTree& tree)
{
    if (allocedTree&&tree.nodes.size())
    {
        Parts = (int*)malloc(tree.primitives.primitivePos.size() * 4 * sizeof(int));
        CreateParts(tree, Parts, 0);
        gpuErrchk(hipMalloc(&devParts, tree.primitives.primitivePos.size() * 4 * sizeof(int)));
        gpuErrchk(hipMemcpy(devParts, Parts, tree.primitives.primitivePos.size() * 4 * sizeof(int), hipMemcpyHostToDevice));
        shapeCount = tree.primitives.primitivePos.size();
        allocedClassicalAdds = true;
    }
}

void Raycaster::CleanUp()
{
    CleanUpTree();
    CleanUpTexture();
    CleanUpClassical();
}

void Raycaster::ChangeAlg(CSGTree& tree, int newAlg)
{

    if (alg == 1)
        CleanUpClassical();
    ChangeTree(tree);
    alg = newAlg;
    if (alg == 0)
    {
        blockDim = dim3(BLOCKXSIZE, BLOCKYSIZE);
    }
    else if (alg == 1)
    {
        blockDim = dim3(BLOCKXSIZERAYMARCH, BLOCKYSIZERAYMARCH);
        SetupClassical(tree);
    }
    else if (alg == 2)
    {
        blockDim = dim3(BLOCKXSIZERAYMARCH, BLOCKYSIZERAYMARCH);
    }
    gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
}

void Raycaster::MapFromCamera(Camera cam)
{
    cudaCamera.position = make_float3(cam.x, cam.y, cam.z);
    cudaCamera.forward = make_float3(cam.forward[0], cam.forward[1], cam.forward[2]);
    cudaCamera.right = make_float3(cam.right[0], cam.right[1], cam.right[2]);
    cudaCamera.up = make_float3(cam.up[0], cam.up[1], cam.up[2]);
    cudaCamera.fov = cam.fov;
}