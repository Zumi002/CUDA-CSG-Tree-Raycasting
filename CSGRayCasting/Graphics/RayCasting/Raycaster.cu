#include "hip/hip_runtime.h"
#include "Raycaster.cuh"
#include "Kernels/Kernels.cuh"

Raycaster::Raycaster()
{
    int sharedMemPerBlock;
    hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(RaycastKernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<32>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<64>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<128>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<256>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<512>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<1024>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(CalculateInterscetionShared<2048>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(RaymarchingKernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));

    gpuErrchk(hipFuncSetAttribute(reinterpret_cast<const void*>(LightningKernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        sharedMemPerBlock));
}

void Raycaster::ChangeTree(CSGTree& tree)
{
    CleanUpTree();
    gpuErrchk(hipMalloc(&cudaTree.nodes, tree.nodes.size() * sizeof(CSGNode)));
    gpuErrchk(hipMalloc(&cudaTree.primitivePos, tree.primitives.primitivePos.size() * sizeof(CudaPrimitivePos)));
    gpuErrchk(hipMalloc(&cudaTree.primitiveColor, tree.primitives.primitiveColor.size() * sizeof(CudaPrimitiveColor)));
    gpuErrchk(hipMalloc(&cudaTree.primitiveParams, tree.primitives.primitiveParameters.size() * sizeof(Parameters)));

    gpuErrchk(hipMemcpy(cudaTree.nodes, tree.nodes.data(), tree.nodes.size() * sizeof(CSGNode), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitivePos, tree.primitives.primitivePos.data(), tree.primitives.primitivePos.size() * sizeof(CudaPrimitivePos), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitiveColor, tree.primitives.primitiveColor.data(), tree.primitives.primitiveColor.size() * sizeof(CudaPrimitiveColor), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(cudaTree.primitiveParams, tree.primitives.primitiveParameters.data(), tree.primitives.primitiveParameters.size() * sizeof(Parameters), hipMemcpyHostToDevice));
    allocedTree = true;
    nodeCount = tree.nodes.size();
    shapeCount = tree.primitives.primitivePos.size();
}

void Raycaster::ChangeSize(int newWidth, int newHeight)
{
    CleanUpTexture();
    width = newWidth;
    height = newHeight;
    gpuErrchk(hipMalloc(&devHits, width * height * sizeof(RayHit)));
    alloced = true;
    CalculateBlockSizes();
}

void Raycaster::ChangeSize(int newWidth, int newHeight, CSGTree& tree)
{
    ChangeSize(newWidth, newHeight);
    ChangeTree(tree);
}

void Raycaster::Raycast(float4* devPBO, Camera cam, DirectionalLight light)
{
    MapFromCamera(cam);
    hipProfilerStart();
    if (alg == 0)
        RaycastKernel << <gridDimSingle, blockDimSingle >> > (cudaCamera, cudaTree.nodes, devBvhNodes, cudaTree.primitivePos, cudaTree.primitiveParams, devHits, width, height);
    else if (alg == 1)
    {
        if (shapeCount <= 32)
        {
            CalculateInterscetionShared<32><< <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 64)
        {
            CalculateInterscetionShared<64> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 128)
        {
            CalculateInterscetionShared<128> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 256)
        {
            CalculateInterscetionShared<256> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 512)
        {
            CalculateInterscetionShared<512> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 1024)
        {
            CalculateInterscetionShared<1024> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
        else if (shapeCount <= 2048)
        {
            CalculateInterscetionShared<2048> << <gridDimClassic, blockDimClassic, 8192 >> > (width, height, shapeCount, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devParts, cudaCamera, devHits);
        }
    }
    else if (alg == 2)
    {
        if (nodeCount <= RAYMARCHSHAREDNODES)
        {
            RaymarchingKernelShared << <gridDimRayMarch, blockDimRayMarch >> > (cudaCamera, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devHits, nodeCount, width, height);
        }
        else
        {
            RaymarchingKernel << <gridDimRayMarch, blockDimRayMarch >> > (cudaCamera, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, devHits, nodeCount, width, height);
        }
    }

    if (collectStats)
    {
        hipMemset(devStats, 0, 2 * sizeof(int));
        PrimitivePerPixelStatistic<<<gridDimRayMarch, blockDimRayMarch>>>(cudaCamera, cudaTree.nodes, cudaTree.primitivePos, cudaTree.primitiveParams, width, height, shapeCount, devStats);
        hipMemcpy(stats, devStats, 2 * sizeof(int), hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipProfilerStop();

    LightningKernel << <gridDimLighting, blockDimLighting >> > (cudaCamera, devHits, cudaTree.primitiveColor, devPBO, light.getLightDir(), width, height);
    hipDeviceSynchronize();
}

void Raycaster::CleanUpTree()
{
    if (allocedTree)
    {
        gpuErrchk(hipFree(cudaTree.nodes));
        gpuErrchk(hipFree(cudaTree.primitivePos));
        gpuErrchk(hipFree(cudaTree.primitiveColor));
        gpuErrchk(hipFree(cudaTree.primitiveParams));
        allocedTree = false;
    }
}
void Raycaster::CleanUpTexture()
{
    if (alloced)
    {
        gpuErrchk(hipFree(devHits));
        alloced = false;
    }
}

void Raycaster::CleanUpClassical()
{
    if (allocedClassicalAdds)
    {
        gpuErrchk(hipFree(devParts));
        free(Parts);
        allocedClassicalAdds = false;
    }
}

void  Raycaster::SetupClassical(CSGTree& tree)
{
    if (allocedTree&&tree.nodes.size())
    {
        Parts = (int*)malloc(tree.primitives.primitivePos.size() * 4 * sizeof(int));
        CreateParts(tree, Parts, 0);
        gpuErrchk(hipMalloc(&devParts, tree.primitives.primitivePos.size() * 4 * sizeof(int)));
        gpuErrchk(hipMemcpy(devParts, Parts, tree.primitives.primitivePos.size() * 4 * sizeof(int), hipMemcpyHostToDevice));
        allocedClassicalAdds = true;
    }
}

void Raycaster::CleanUp()
{
    CleanUpTree();
    CleanUpTexture();
    CleanUpClassical();
    CleanUpSingleHit();

    if (collectStats)
    {
        gpuErrchk(hipFree(devStats));
        delete[] stats;
    }
}

void Raycaster::ChangeAlg(CSGTree& tree, int newAlg)
{
    if (alg == 0)
    {
        CleanUpSingleHit();
    }
    else if (alg == 1)
    {
        CleanUpClassical();
    }

    ChangeTree(tree);
    alg = newAlg;

    if (alg == 0)
    {
        SetupSingleHit(tree);
    }
    else if (alg == 1)
    {
        SetupClassical(tree);
    }
    CalculateBlockSizes();
}



void Raycaster::MapFromCamera(Camera cam)
{
    cudaCamera.position = make_float3(cam.x, cam.y, cam.z);
    cudaCamera.forward = make_float3(cam.forward[0], cam.forward[1], cam.forward[2]);
    cudaCamera.right = make_float3(cam.right[0], cam.right[1], cam.right[2]);
    cudaCamera.up = make_float3(cam.up[0], cam.up[1], cam.up[2]);
    cudaCamera.fov = cam.fov;
}

void Raycaster::CalculateBlockSizes()
{
    blockDimSingle = dim3(BLOCKXSIZE, BLOCKYSIZE);
    gridDimSingle = dim3((width + blockDimSingle.x - 1) / blockDimSingle.x, (height + blockDimSingle.y - 1) / blockDimSingle.y);
    blockDimClassic = dim3(BLOCKXSIZECLASSIC, BLOCKYSIZECLASSIC);
    gridDimClassic = dim3((width + blockDimClassic.x - 1) / blockDimClassic.x, (height + blockDimClassic.y - 1) / blockDimClassic.y);
    blockDimRayMarch = dim3(BLOCKXSIZERAYMARCH, BLOCKYSIZERAYMARCH);
    gridDimRayMarch = dim3((width + blockDimRayMarch.x - 1) / blockDimRayMarch.x, (height + blockDimRayMarch.y - 1) / blockDimRayMarch.y);
    blockDimLighting = dim3(BLOCKXSIZELIGHTING, BLOCKYSIZELIGHTING);
    gridDimLighting = dim3((width + blockDimLighting.x - 1) / blockDimLighting.x, (height + blockDimLighting.y - 1) / blockDimLighting.y);
}

void Raycaster::SetupSingleHit(CSGTree& tree)
{
    if (allocedTree && tree.nodes.size())
    {
        std::vector<BVHNode> bvhNodes = tree.ConstructBVH();
        gpuErrchk(hipMalloc(&devBvhNodes, bvhNodes.size()*sizeof(BVHNode)));
        gpuErrchk(hipMemcpy(devBvhNodes, bvhNodes.data(), bvhNodes.size() * sizeof(BVHNode), hipMemcpyHostToDevice));
        allocedSingleHitAdds = true;
    }
}

void Raycaster::CleanUpSingleHit()
{
    if (allocedSingleHitAdds)
    {
        gpuErrchk(hipFree(devBvhNodes));
        allocedSingleHitAdds = false;
    }
}

void Raycaster::CollectStats()
{
    collectStats = true;
    gpuErrchk(hipMalloc(&devStats, 2 * sizeof(int)));
    stats = new int[2];
}