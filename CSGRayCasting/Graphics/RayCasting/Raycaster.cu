#include "hip/hip_runtime.h"
#include "Raycaster.cuh"

void Raycaster::ChangeSize(int newWidth, int newHeight, CSGTree tree)
{
	CleanUp();

	width = newWidth;
	height = newHeight;

	blockDim = dim3(BLOCKXSIZE, BLOCKYSIZE);
	gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

	gpuErrchk(hipMalloc(&cudaTree.nodes, tree.nodes.size() * sizeof(CSGNode)));
	gpuErrchk(hipMalloc(&cudaTree.primitives, tree.primitives.primitives.size() * sizeof(Primitive)));

	gpuErrchk(hipMemcpy(cudaTree.nodes, tree.nodes.data(), tree.nodes.size() * sizeof(CSGNode), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(cudaTree.primitives, tree.primitives.primitives.data(), tree.primitives.primitives.size() * sizeof(Primitive), hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&devHits, width * height * sizeof(RayHit)));
	alloced = true;
}

void Raycaster::Raycast(float4* devPBO, Camera cam, DirectionalLight light)
{

	RaycastKernel<<<gridDim, blockDim>>>(cam, cudaTree, devHits, width, height);
	hipDeviceSynchronize();

	

	LightningKernel<<<gridDim, blockDim>>>(cam, devHits, cudaTree.primitives, devPBO, light.getLightDir(), width, height);
	hipDeviceSynchronize();
	
}

void Raycaster::CleanUp()
{
	if (alloced)
	{
		gpuErrchk(hipFree(devHits));
		gpuErrchk(hipFree(cudaTree.nodes));
		gpuErrchk(hipFree(cudaTree.primitives));
		alloced = false;
	}
}