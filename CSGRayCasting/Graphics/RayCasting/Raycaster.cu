#include "hip/hip_runtime.h"
#include "Raycaster.cuh"

void Raycaster::ChangeSize(int newWidth, int newHeight, CSGTree tree)
{
	if (alloced)
	{
		gpuErrchk(hipFree(devHits));
		gpuErrchk(hipFree(cudaTree.nodes));
		gpuErrchk(hipFree(cudaTree.primitives));
	}

	width = newWidth;
	height = newHeight;

	blockDim = dim3(BLOCKXSIZE, BLOCKYSIZE);
	gridDim = dim3((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

	gpuErrchk(hipMalloc(&cudaTree.nodes, tree.nodes.size() * sizeof(CSGNode)));
	gpuErrchk(hipMalloc(&cudaTree.primitives, tree.primitives.primitives.size() * sizeof(Primitive)));

	gpuErrchk(hipMemcpy(cudaTree.nodes, tree.nodes.data(), tree.nodes.size() * sizeof(CSGNode), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(cudaTree.primitives, tree.primitives.primitives.data(), tree.primitives.primitives.size() * sizeof(Primitive), hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&devHits, width * height * sizeof(RayHit)));
	alloced = true;
}

void Raycaster::Raycast(float4* devPBO, Camera cam)
{

	RaycastKernel<<<gridDim, blockDim>>>(cam, cudaTree, devHits, width, height);
	hipDeviceSynchronize();

	LightningKernel<<<gridDim, blockDim>>>(cam, devHits, cudaTree.primitives, devPBO, width, height);
	hipDeviceSynchronize();
	
}

