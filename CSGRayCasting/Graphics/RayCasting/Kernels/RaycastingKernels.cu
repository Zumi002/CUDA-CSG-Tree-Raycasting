#include "hip/hip_runtime.h"
#include "RaycastingKernels.cuh"

__global__ void RaycastKernel(Camera cam, CudaCSGTree tree, RayHit* hits, float width, float height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	// Calculate normalized device coordinates (NDC)
	float u = (float)x / (width - 1);
	float v = (float)y / (height - 1);

	// Convert to screen space coordinates (-1 to 1)
	float nx = (2.0f * u - 1.0f) * (width / height) * tan(cam.fov / 2.0f);
	float ny = (1.0f - 2.0f * v) * tan(cam.fov / 2.0f);


	// Create ray from camera
	float3 rayOrigin = make_float3(cam.x, cam.y, cam.z);
	float3 rayDirection = normalize(make_float3(
		cam.right[0] * nx + cam.up[0] * ny + cam.forward[0],
		cam.right[1] * nx + cam.up[1] * ny + cam.forward[1],
		cam.right[2] * nx + cam.up[2] * ny + cam.forward[2]
	));

	Ray ray(rayOrigin, rayDirection);
	RayHitMinimal hitInfo;

	// For now, just test against the first sphere
	
	CSGRayCast(tree, ray, hitInfo);
	int pixelIdx = (y * (int)width + x);
	
	RayHit detailedHitInfo;
	detailedHitInfo.hit = false;
	if (hitInfo.hit != CSG::CSGRayHit::Miss)
	{
		if(hitInfo.primitiveType == CSGTree::NodeType::Sphere)
			sphereHitDetails(ray, tree.primitives[hitInfo.primitiveIdx], hitInfo, detailedHitInfo);
	}
	hits[pixelIdx] = detailedHitInfo;
}

__global__ void LightningKernel(Camera cam, RayHit* hits, Primitive* primitives, float4* output, float width, float height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	int pixelIdx = (y * (int)width + x);

	RayHit hitInfo = hits[pixelIdx];



	if (hitInfo.hit)
	{
		float3 color = make_float3(primitives[hitInfo.primitiveIdx].r,
			primitives[hitInfo.primitiveIdx].g,
			primitives[hitInfo.primitiveIdx].b);
		float3 lightPos = make_float3(-3.0f, 5.0f, 0);  // Light position
		float3 lightColor = make_float3(1.0f, 1.0f, 1.0f); // White light

		// Material properties
		float ka = 0.2f;    // Ambient intensity
		float kd = 0.8f;    // Diffuse intensity
		float ks = 0.7f;    // Specular intensity
		float shininess = 30.0f; // Specular shininess

		// Calculate lighting vectors

		float3 lightDir = normalize(lightPos - hitInfo.position);
		float3 viewDir = normalize(make_float3(cam.x, cam.y, cam.z) - hitInfo.position);
		float3 reflectDir = reflect(-lightDir, hitInfo.normal);

		// Ambient component
		float3 ambient = ka * lightColor;

		// Diffuse component
		float diff = max(dot(hitInfo.normal, lightDir), 0.0f);
		float3 diffuse = kd * diff * lightColor;

		// Specular component
		float spec = pow(max(dot(viewDir, reflectDir), 0.0f), shininess);
		float3 specular = ks * spec * lightColor;

		// Combine all components
		float3 finalColor = make_float3(
			color.x * (ambient.x + diffuse.x + specular.x),
			color.y * (ambient.y + diffuse.y + specular.y),
			color.z * (ambient.z + diffuse.z + specular.z)
		);

		// Clamp colors to [0,1]
		finalColor.x = fmin(fmax(finalColor.x, 0.0f), 1.0f);
		finalColor.y = fmin(fmax(finalColor.y, 0.0f), 1.0f);
		finalColor.z = fmin(fmax(finalColor.z, 0.0f), 1.0f);

		output[pixelIdx] = make_float4(finalColor.x, finalColor.y, finalColor.z, 1.0f);
	}
	else
	{
		output[pixelIdx] = make_float4(0.08f, 0.08f, 0.11f, 1);
	}
}


__device__ bool sphereHit(const Ray& ray, const Primitive& sphere, RayHitMinimal& hitInfo, float& tmin)
{
	hitInfo.hit = CSG::CSGRayHit::Miss;
	hitInfo.primitiveIdx = sphere.id;
	float3 oc = make_float3(
		ray.origin.x - sphere.x,
		ray.origin.y - sphere.y,
		ray.origin.z - sphere.z
	);

	float a = dot(ray.direction, ray.direction);
	float b = dot(oc, ray.direction);
	float c = dot(oc, oc) - sphere.params.sphereParameters.radius * sphere.params.sphereParameters.radius;
	float discriminant = b * b - a * c;

	if (discriminant < 0) return false;

	float temp = (-b - sqrtf(discriminant)) / (a);
	if (temp <= tmin) {
		temp = (-b + sqrtf(discriminant)) / (a);
		if (temp <= tmin)
		{
			hitInfo.t = -1;
			hitInfo.hit = CSG::CSGRayHit::Miss;
			hitInfo.primitiveIdx = -1;
			return false;
		}
	}

	hitInfo.t = temp;

	float3 normal = ray.computePosition(temp);

	normal =
		make_float3(
			normal.x - sphere.x,
			normal.y - sphere.y,
			normal.z - sphere.z);
	
	if (dot(normal, ray.direction) <= 0)
		hitInfo.hit = CSG::CSGRayHit::Enter;
	else
		hitInfo.hit = CSG::CSGRayHit::Exit;

	hitInfo.primitiveType = CSGTree::NodeType::Sphere;

	return true;
}

__device__ void sphereHitDetails(const Ray& ray, const Primitive& sphere, const RayHitMinimal& hitInfo, RayHit& detailedHitInfo)
{
	detailedHitInfo.hit = true;
	detailedHitInfo.t = hitInfo.t;
	detailedHitInfo.position = ray.computePosition(detailedHitInfo.t);
	detailedHitInfo.primitiveIdx = hitInfo.primitiveIdx;
	detailedHitInfo.normal = normalize(
		make_float3(
			detailedHitInfo.position.x - sphere.x,
			detailedHitInfo.position.y - sphere.y,
			detailedHitInfo.position.z - sphere.z
		)
	);
	if (hitInfo.hit & CSG::CSGRayHit::Flip)
		detailedHitInfo.normal = -detailedHitInfo.normal;
	if (hitInfo.hit & CSG::CSGRayHit::Exit)
		detailedHitInfo.normal = -detailedHitInfo.normal;
}

__device__ void CSGRayCast(CudaCSGTree& tree, Ray& ray, RayHitMinimal& resultRayhit)
{

	CudaStack<unsigned char, MAXSTACKSIZE> actionStack;
	CudaStack<RayHitMinimal, MAXSTACKSIZE> primitiveStack;
	CudaStack<float, MAXSTACKSIZE> timeStack;

	float tmin = 0;
	CSGNode node = CSGNode(0, 0, 0, 0, 0);
	RayHitMinimal leftRay;
	RayHitMinimal rightRay;
	actionStack.push(CSG::CSGActions::Compute);
	unsigned char action = CSG::CSGActions::GotoLft;
	bool run = true;

	while (run||actionStack.size()>0)
	{
		if (action & CSG::CSGActions::SaveLft)
		{
			tmin = timeStack.pop();
			primitiveStack.push(leftRay);
			action = CSG::CSGActions::GotoRgh;
		}
		if (action & (CSG::CSGActions::GotoLft | CSG::CSGActions::GotoRgh))
		{
			GoTo(actionStack,
				 primitiveStack,
				 timeStack,
				 action,
				 node,
				 tree,
				 leftRay,
				 rightRay,
				 ray,
				 tmin, 
				 run);
		}
		if (action & (CSG::CSGActions::LoadLft | CSG::CSGActions::LoadRgh | CSG::CSGActions::Compute))
		{
			Compute(actionStack,
				primitiveStack,
				timeStack,
				action,
				node,
				tree,
				leftRay,
				rightRay,
				tmin,
				run);
		}
	}

	resultRayhit = leftRay;
}

__device__ void GoTo(
	CudaStack<unsigned char, MAXSTACKSIZE>& actionStack,
	CudaStack<RayHitMinimal, MAXSTACKSIZE>& primitiveStack,
	CudaStack<float, MAXSTACKSIZE>& timeStack,
	unsigned char& action,
	CSGNode& node,
	CudaCSGTree& tree,
	RayHitMinimal& leftRay,
	RayHitMinimal& rightRay,
	Ray& ray,
	float& tmin,
	bool& run)
{
	if (action & CSG::CSGActions::GotoLft)
	{
		node = tree.nodes[node.left];
	}
	else
	{
		node = tree.nodes[node.right];
	}

	if (node.type == CSGTree::NodeType::Union ||
		node.type == CSGTree::NodeType::Difference ||
		node.type == CSGTree::NodeType::Intersection)
	{
		bool gotoL = true;
		bool gotoR = true;
		if (gotoL && (tree.nodes[node.left].primitiveIdx != -1))
		{
			sphereHit(ray, tree.primitives[tree.nodes[node.left].primitiveIdx], leftRay, tmin);
			gotoL = false;
		}
		if (gotoR && (tree.nodes[node.right].primitiveIdx != -1))
		{
			sphereHit(ray, tree.primitives[tree.nodes[node.right].primitiveIdx], rightRay, tmin);
			gotoR = false;
		}
		if (gotoL || gotoR)
		{
			if (!gotoL)
			{
				primitiveStack.push(leftRay);
				actionStack.push(CSG::CSGActions::LoadLft);
				action = CSG::CSGActions::GotoRgh;
			}
			else if (!gotoR)
			{
				primitiveStack.push(rightRay);
				actionStack.push(CSG::CSGActions::LoadRgh);
				action = CSG::CSGActions::GotoLft;
			}
			else
			{
				timeStack.push(tmin);
				actionStack.push(CSG::CSGActions::LoadLft);
				actionStack.push(CSG::CSGActions::SaveLft);
				action = CSG::CSGActions::GotoLft;
			}
		}
		else
		{
			action = CSG::CSGActions::Compute;
		}

	}
	else
	{
		if (action & CSG::CSGActions::GotoLft)
		{
			sphereHit(ray, tree.primitives[node.primitiveIdx], leftRay, tmin);
		}
		else
		{
			sphereHit(ray, tree.primitives[node.primitiveIdx], rightRay, tmin);
		}
		action = actionStack.pop();
		node = GetParent(tree, node, run);
	}
}

__device__ void Compute(
	CudaStack<unsigned char, MAXSTACKSIZE>& actionStack,
	CudaStack<RayHitMinimal, MAXSTACKSIZE>& primitiveStack,
	CudaStack<float, MAXSTACKSIZE>& timeStack,
	unsigned char& action,
	CSGNode& node,
	CudaCSGTree& tree,
	RayHitMinimal& leftRay,
	RayHitMinimal& rightRay,
	float& tmin, 
	bool& run)
{
	if (action & (CSG::CSGActions::LoadLft | CSG::CSGActions::LoadRgh))
	{
		if (action & CSG::CSGActions::LoadLft)
		{
			leftRay = primitiveStack.pop();
		}
		else
		{
			rightRay = primitiveStack.pop();
		}
	}
	int actions = LookUpActions(leftRay.hit, rightRay.hit, node.type);
	if ((actions & CSG::HitActions::RetL) || ((actions & CSG::HitActions::RetLIfCloser) && (leftRay.t <= rightRay.t)))
	{
		rightRay = leftRay;
		action = actionStack.pop();
		node = GetParent(tree, node, run);
	}
	else if ((actions & CSG::HitActions::RetR) || ((actions & CSG::HitActions::RetRIfCloser) && (leftRay.t > rightRay.t)))
	{
		if (actions & CSG::HitActions::FlipR)
		{
			rightRay.hit ^= CSG::CSGRayHit::Flip;
			rightRay.hit ^= CSG::CSGRayHit::Exit;
			rightRay.hit ^= CSG::CSGRayHit::Enter;
			
		}
		leftRay = rightRay;
		action = actionStack.pop();
		node = GetParent(tree, node, run);
	}
	else if ((actions & CSG::HitActions::LoopL) || ((actions & CSG::HitActions::LoopLIfCloser) && (leftRay.t <= rightRay.t)))
	{
		tmin = leftRay.t;
		primitiveStack.push(rightRay);
		actionStack.push(CSG::CSGActions::LoadRgh);
		action = CSG::CSGActions::GotoLft;
	}
	else if ((actions & CSG::HitActions::LoopR) || ((actions & CSG::HitActions::LoopRIfCloser) && (leftRay.t > rightRay.t)))
	{
		tmin = rightRay.t;
		primitiveStack.push(leftRay);
		actionStack.push(CSG::CSGActions::LoadLft);
		action = CSG::CSGActions::GotoRgh;
	}
	else
	{
		rightRay = RayHitMinimal();
		leftRay = RayHitMinimal();
		action = actionStack.pop();
		node = GetParent(tree, node, run);
	}
}


__device__ int LookUpActions(unsigned char lHit, unsigned char rHit, int op)
{
	static int unionTable[3][3] = {
	{{CSG::HitActions::RetLIfCloser | CSG::HitActions::RetRIfCloser},{CSG::HitActions::RetRIfCloser | CSG::HitActions::LoopL},{CSG::HitActions::RetL}},
	{{CSG::HitActions::RetLIfCloser | CSG::HitActions::LoopR},{CSG::HitActions::LoopLIfCloser | CSG::HitActions::LoopRIfCloser},{CSG::HitActions::RetL}},
	{{CSG::HitActions::RetR},{CSG::HitActions::RetR},{CSG::HitActions::MissAction}} };
	static int intersectionTable[3][3] = {
	{{CSG::HitActions::LoopLIfCloser | CSG::HitActions::LoopRIfCloser},{CSG::HitActions::RetLIfCloser|CSG::HitActions::LoopR},{CSG::HitActions::MissAction}},
	{{CSG::HitActions::RetRIfCloser | CSG::HitActions::LoopL},{CSG::HitActions::RetLIfCloser | CSG::HitActions::RetRIfCloser},{CSG::HitActions::MissAction}},
	{{CSG::HitActions::MissAction},{CSG::HitActions::MissAction},{CSG::HitActions::MissAction}} };
	static int differenceTable[3][3] = {
	{{CSG::HitActions::RetLIfCloser | CSG::HitActions::LoopR},{CSG::HitActions::LoopLIfCloser | CSG::HitActions::LoopRIfCloser},{CSG::HitActions::RetL}},
	{{CSG::HitActions::RetLIfCloser | CSG::HitActions::RetRIfCloser |CSG::HitActions::FlipR},{CSG::HitActions::RetRIfCloser | CSG::HitActions::FlipR|CSG::HitActions::LoopL},{CSG::HitActions::RetL}},
	{{CSG::HitActions::MissAction},{CSG::HitActions::MissAction},{CSG::HitActions::MissAction}} };

	if (lHit & CSG::CSGRayHit::Enter)
		lHit = 0;
	if (lHit & CSG::CSGRayHit::Exit)
		lHit = 1;
	if (lHit & CSG::CSGRayHit::Miss)
		lHit = 2;

	if (rHit & CSG::CSGRayHit::Enter)
		rHit = 0;
	if (rHit & CSG::CSGRayHit::Exit)
		rHit = 1;
	if (rHit & CSG::CSGRayHit::Miss)
		rHit = 2;

	if (op == CSGTree::NodeType::Union)
	{
		return unionTable[lHit][rHit];
	}
	if (op == CSGTree::NodeType::Intersection)
	{
		return intersectionTable[lHit][rHit];
	}
	if (op == CSGTree::NodeType::Difference)
	{
		return differenceTable[lHit][rHit];
	}
	return -1;
}

__device__ CSGNode GetParent(CudaCSGTree& tree, CSGNode& node, bool& run)
{
	if (node.parent >= 0)
	{
		return tree.nodes[node.parent];
	}
	run = false;
	return CSGNode(0, 0, 0, 0, 0);
}